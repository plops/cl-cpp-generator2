#include "hip/hip_runtime.h"
// david kirk: programming massively parallel processors (third ed) p. 175
// prefix sum
#include <cassert>
#include <chrono>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
auto g_start = static_cast<typeof(
    std::chrono::high_resolution_clock::now().time_since_epoch().count())>(0);
void sequential_scan(float *x, float *y, int n) {
  auto accum = x[0];
  y[0] = accum;
  for (int i = 1; i < n; (i)++) {
    (accum) += (x[i]);
    y[i] = accum;
  };
}
enum { SECTION_SIZE = 8 };
__global__ void kogge_stone_scan_kernel(float *x, float *y, int n) {
  __shared__ float XY[SECTION_SIZE];
  auto i = ((threadIdx.x) + (((blockDim.x) * (blockIdx.x))));
  if (i < n) {
    XY[threadIdx.x] = x[i];
  };
  for (int stride = 1; stride < blockDim.x; stride = ((2) * (stride))) {
    __syncthreads();
    if ((stride) <= (threadIdx.x)) {
      (XY[threadIdx.x]) += (XY[((threadIdx.x) - (stride))]);
    };
  }
  y[i] = XY[threadIdx.x];
};
int main() {
  g_start =
      std::chrono::high_resolution_clock::now().time_since_epoch().count();
  auto n_cuda = 0;
  {
    auto r = hipGetDeviceCount(&n_cuda);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (g_start)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipGetDeviceCount(&n_cuda) => ") << (r)
                << (" '") << (hipGetErrorString(r)) << ("' ") << (" n_cuda=")
                << (n_cuda) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipSetDevice(0);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (g_start)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipSetDevice(0) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  float x[8] = {3, 1, 7, 0, 4, 1, 6, 3};
  float y[8];
  sequential_scan(x, y, ((sizeof(x)) / (sizeof(*x))));
  for (int i = 0; i < ((sizeof(x)) / (sizeof(*x))); (i) += (1)) {
    (std::cout) << (y[i]) << (std::endl);
  };
  return 0;
}