#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <hip/hip_runtime.h>
using namespace std;
__global__ void matrix_mul(int *a, int *b, int *c, int n) {
  int col = ((((blockDim.x) * (blockIdx.x))) + (threadIdx.x));
  int row = ((((blockDim.y) * (blockIdx.y))) + (threadIdx.y));
  int sum = 0;
  if (((row < n) && (col < n))) {
    for (int k = 0; k < n; (k) += (1)) {
      (temp_sum) +=
          (((a[((k) + (((row) * (n))))]) * (b[((col) + (((k) * (n))))])));
    }
    c[((col) + (((row) * (n))))] = temp_sum;
  };
}
void init_matrix(int *a, int n) {
  for (int i = 0; i < ((n) * (n)); (i) += (1)) {
    a[i] = rand() % 100;
  }
}
int main() {
  // 1024x1024 square matrix
  auto n = 1 << 10;
  auto bytes = ((n) * (n) * (sizeof(int)));
  int *a;
  int *b;
  int *c;
  hipMallocManaged(&(a), bytes);
  hipMallocManaged(&(b), bytes);
  hipMallocManaged(&(c), bytes);
  init_matrix(a, n);
  init_matrix(b, n);
  // one thread per output element
  // square thread blocks
  auto threads = 16;
  auto blocks = ((((n) + (((threads) - (1))))) / (threads));
  // n=1024 threads=14 blocks=1037/14=74
  // n=1024 threads=16 blocks=1039/16=64
  // n=1024 threads=32 blocks=1055/32=32
  // kernel launch parameters
  auto threads2 = dim3(threads, threads);
  auto blocks2 = dim3(blocks, blocks);
  // async kernel start
  matrix_mul<<<blocks2, threads2, 0, 0>>>(a, b, c, n);
  // managed memory need explicit sync
  hipDeviceSynchronize();
  vector_add_cpu_assert(a, b, c, n);
  return 0;
}