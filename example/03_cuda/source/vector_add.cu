#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void vector_add() {}
void init_array(int *a, int n) {
  for (int i = 0; i < n; (i) += (1)) {
    a[i] = rand() % 100;
  }
}
int main() {
  int n = 1 << 20;
  size_t bytes = ((n) * (sizeof(bytes)));
  int *a;
  int *b;
  int *c;
  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);
  init_array(a, n);
  init_array(b, n);
}