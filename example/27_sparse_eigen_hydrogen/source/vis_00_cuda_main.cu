#include "hip/hip_runtime.h"

#include "utils.h"

#include "globals.h"

;
#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <experimental/iterator>
#include <fstream>
#include <iostream>
#include <string>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "arpack-ng/ICB/arpack.hpp"

using namespace std::chrono_literals;
State state = {};
__global__ void kernel_hamiltonian(float *out, float *in) {
  auto idx = ((((blockIdx.x) * (blockDim.x))) + (threadIdx.x));
  auto ri = ((idx) * ((5.00e-2)));
  auto l = 0;
  auto Z = 1;
  if ((idx) < (1000)) {
    auto Vr = ((((((l) * (((l) + (1))))) / (((ri) * (ri))))) -
               (((((2) * (Z))) / (ri))));
    if ((((1) <= (idx)) && ((idx) <= (998)))) {
      out[idx] = ((((((1) / ((2.50e-3)))) *
                    (((in[((idx) - (1))]) + (in[((idx) + (1))]))))) +
                  (((((((-2) / ((2.50e-3)))) + (Vr))) * (in[idx]))));
    };
  };
}
int main(int argc, char const *const *const argv) {
  state._main_version = "fff5b020b8e56c2ddbac7efcb13629e0d8344782";
  state._code_repository = "https://github.com/plops/cl-cpp-generator2/tree/"
                           "master/example/27_sparse_eigen_hydrogen";
  state._code_generation_time = "14:50:59 of Sunday, 2020-07-12 (GMT+1)";
  state._start_time =
      std::chrono::high_resolution_clock::now().time_since_epoch().count();

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("start main") << (" ")
      << (std::setw(8)) << (" state._main_version='") << (state._main_version)
      << ("'") << (std::endl) << (std::flush);

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("") << (" ")
      << (std::setw(8)) << (" state._code_repository='")
      << (state._code_repository) << ("'") << (std::endl) << (std::flush);

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("") << (" ")
      << (std::setw(8)) << (" state._code_generation_time='")
      << (state._code_generation_time) << ("'") << (std::endl) << (std::flush);
  hipStream_t stream;
  {
    auto res = hipStreamCreate(&stream);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipStreamCreate(&stream)");
    };
  };
  float *in;
  float *out;
  {
    auto res = hipMallocManaged(&in, ((1000) * (sizeof(float))));
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error(
          "hipMallocManaged(&in, ((1000)*(sizeof(float))))");
    };
  };
  {
    auto res = hipMallocManaged(&out, ((1000) * (sizeof(float))));
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error(
          "hipMallocManaged(&out, ((1000)*(sizeof(float))))");
    };
  };
  kernel_hamiltonian<<<2, 512, 0, stream>>>(out, in);
  hipStreamSynchronize(stream);
  {
    auto res = hipFree(out);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipFree(out)");
    };
  };
  {
    auto res = hipFree(in);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipFree(in)");
    };
  };

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("end main") << (" ")
      << (std::endl) << (std::flush);
  return 0;
};