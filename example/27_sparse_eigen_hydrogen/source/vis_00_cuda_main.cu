#include "hip/hip_runtime.h"

#include "utils.h"

#include "globals.h"

;
#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <experimental/iterator>
#include <fstream>
#include <iostream>
#include <string>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "arpack-ng/ICB/arpack.hpp"
#include "arpackpp/include/arrssym.h"

using namespace std::chrono_literals;
State state = {};
__global__ void kernel_hamiltonian(float *out, float *in) {
  auto idx = ((((blockIdx.x) * (blockDim.x))) + (threadIdx.x));
  auto ri = ((idx) * ((5.00e-2)));
  auto l = 0;
  auto Z = 1;
  if ((idx) < (1000)) {
    auto Vr = ((((((l) * (((l) + (1))))) / (((ri) * (ri))))) -
               (((((2) * (Z))) / (ri))));
    if ((((1) <= (idx)) && ((idx) <= (998)))) {
      out[idx] = ((((((1) / ((2.50e-3)))) *
                    (((in[((idx) - (1))]) + (in[((idx) + (1))]))))) +
                  (((((((-2) / ((2.50e-3)))) + (Vr))) * (in[idx]))));
    } else {
      if ((idx) == (0)) {
        out[idx] = ((((((1) / ((2.50e-3)))) * (((in[((idx) + (1))]))))) +
                    (((((((-2) / ((2.50e-3)))) + (Vr))) * (in[idx]))));
      } else {
        out[idx] = ((((((1) / ((2.50e-3)))) * (((in[((idx) - (1))]))))) +
                    (((((((-2) / ((2.50e-3)))) + (Vr))) * (in[idx]))));
      }
    };
  };
}
int main(int argc, char const *const *const argv) {
  state._main_version = "4b36f2cbf4824fc2d9155499a6684edc1e431fc7";
  state._code_repository = "https://github.com/plops/cl-cpp-generator2/tree/"
                           "master/example/27_sparse_eigen_hydrogen";
  state._code_generation_time = "15:27:36 of Sunday, 2020-07-12 (GMT+1)";
  state._start_time =
      std::chrono::high_resolution_clock::now().time_since_epoch().count();

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("start main") << (" ")
      << (std::setw(8)) << (" state._main_version='") << (state._main_version)
      << ("'") << (std::endl) << (std::flush);

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("") << (" ")
      << (std::setw(8)) << (" state._code_repository='")
      << (state._code_repository) << ("'") << (std::endl) << (std::flush);

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("") << (" ")
      << (std::setw(8)) << (" state._code_generation_time='")
      << (state._code_generation_time) << ("'") << (std::endl) << (std::flush);
  hipStream_t stream;
  {
    auto res = hipStreamCreate(&stream);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipStreamCreate(&stream)");
    };
  };
  float *in;
  float *out;
  {
    auto res = hipMallocManaged(&in, ((1000) * (sizeof(float))));
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error(
          "hipMallocManaged(&in, ((1000)*(sizeof(float))))");
    };
  };
  {
    auto res = hipMallocManaged(&out, ((1000) * (sizeof(float))));
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error(
          "hipMallocManaged(&out, ((1000)*(sizeof(float))))");
    };
  };
  auto prob = ARrcSymStdEig<float>(1000, 4L);
  while (!(prob.ArnoldiBasisFound())) {
    prob.TakeStep();
    auto ido = prob.GetIdo();
    if ((((ido) == (1)) || ((ido) == (-1)))) {
      auto in_ = prob.GetVector();
      auto out_ = prob.PutVector();
      // multiply
      for (auto i = 0; (i) < (1000); (i) += (1)) {
        in[i] = in_[i];
      }
      kernel_hamiltonian<<<2, 512, 0, stream>>>(out, in);
      hipStreamSynchronize(stream);
      for (auto i = 0; (i) < (1000); (i) += (1)) {
        out_[i] = out[i];
      };
    };
  }
  prob.FindEigenvectors();
  {
    auto res = hipFree(out);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipFree(out)");
    };
  };
  {
    auto res = hipFree(in);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipFree(in)");
    };
  };

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("end main") << (" ")
      << (std::endl) << (std::flush);
  return 0;
};