#include "hip/hip_runtime.h"

#include "utils.h"

#include "globals.h"

;
#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <experimental/iterator>
#include <fstream>
#include <iostream>
#include <string>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "arpack-ng/ICB/arpack.hpp"
#include "arpackpp/include/arrssym.h"

using namespace std::chrono_literals;
State state = {};
__global__ void kernel_hamiltonian(float *out, float *in) {
  auto idx = ((((blockIdx.x) * (blockDim.x))) + (threadIdx.x));
  auto ri = ((((1) + (idx))) * ((1.6666667e-2)));
  auto l = 0;
  auto Z = 1;
  if ((idx) < (3000)) {
    auto Vr = ((((((l) * (((l) + (1))))) / (((ri) * (ri))))) -
               (((((2) * (Z))) / (ri))));
    if ((((1) <= (idx)) && ((idx) <= (2998)))) {
      out[idx] = ((((((-1) / ((2.777778e-4)))) *
                    (((in[((idx) - (1))]) + (in[((idx) + (1))]))))) +
                  (((((((2) / ((2.777778e-4)))) + (Vr))) * (in[idx]))));
    } else {
      if ((idx) == (0)) {
        out[idx] = ((((((-1) / ((2.777778e-4)))) * (((in[((idx) + (1))]))))) +
                    (((((((2) / ((2.777778e-4)))) + (Vr))) * (in[idx]))));
      } else {
        out[idx] = ((((((-1) / ((2.777778e-4)))) * (((in[((idx) - (1))]))))) +
                    (((((((2) / ((2.777778e-4)))) + (Vr))) * (in[idx]))));
      }
    };
  };
}
int main(int argc, char const *const *const argv) {
  state._main_version = "c3ac14d0ff3c0ed5c9a6c5929c9b71c411c8ea8d";
  state._code_repository = "https://github.com/plops/cl-cpp-generator2/tree/"
                           "master/example/27_sparse_eigen_hydrogen";
  state._code_generation_time = "16:22:13 of Sunday, 2020-07-12 (GMT+1)";
  state._start_time =
      std::chrono::high_resolution_clock::now().time_since_epoch().count();

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("start main") << (" ")
      << (std::setw(8)) << (" state._main_version='") << (state._main_version)
      << ("'") << (std::endl) << (std::flush);

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("") << (" ")
      << (std::setw(8)) << (" state._code_repository='")
      << (state._code_repository) << ("'") << (std::endl) << (std::flush);

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("") << (" ")
      << (std::setw(8)) << (" state._code_generation_time='")
      << (state._code_generation_time) << ("'") << (std::endl) << (std::flush);
  hipStream_t stream;
  auto blocks = 6;
  auto threads = 512;
  {
    auto res = hipStreamCreate(&stream);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipStreamCreate(&stream)");
    };
  };
  float *in;
  float *out;
  {
    auto res = hipMallocManaged(&in, ((3000) * (sizeof(float))));
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error(
          "hipMallocManaged(&in, ((3000)*(sizeof(float))))");
    };
  };
  {
    auto res = hipMallocManaged(&out, ((3000) * (sizeof(float))));
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error(
          "hipMallocManaged(&out, ((3000)*(sizeof(float))))");
    };
  };
  // relevant arpack++ example
  // https://github.com/m-reuter/arpackpp/blob/master/examples/reverse/sym/rsymreg.cc
  ;
  // The following values of which are available:
  // which = 'LM' : Eigenvalues with largest magnitude (eigs, eigsh), that is,
  // largest eigenvalues in the euclidean norm of complex numbers. which = 'SM'
  // : Eigenvalues with smallest magnitude (eigs, eigsh), that is, smallest
  // eigenvalues in the euclidean norm of complex numbers. which = 'LR' :
  // Eigenvalues with largest real part (eigs). which = 'SR' : Eigenvalues with
  // smallest real part (eigs). which = 'LI' : Eigenvalues with largest
  // imaginary part (eigs). which = 'SI' : Eigenvalues with smallest imaginary
  // part (eigs). which = 'LA' : Eigenvalues with largest algebraic value
  // (eigsh), that is, largest eigenvalues inclusive of any negative sign. which
  // = 'SA' : Eigenvalues with smallest algebraic value (eigsh), that is,
  // smallest eigenvalues inclusive of any negative sign. which = 'BE' :
  // Eigenvalues from both ends of the spectrum (eigsh). Note that ARPACK is
  // generally better at finding extremal eigenvalues, that is, eigenvalues with
  // large magnitudes. In particular, using which = 'SM' may lead to slow
  // execution time and/or anomalous results. A better approach is to use
  // shift-invert mode.
  ;
  auto prob = ARrcSymStdEig<float>(3000, 4L, "SA", 0, (0.f), 100000);
  while (!(prob.ArnoldiBasisFound())) {
    prob.TakeStep();
    auto ido = prob.GetIdo();
    if ((((ido) == (1)) || ((ido) == (-1)))) {
      auto in_ = prob.GetVector();
      auto out_ = prob.PutVector();
      // multiply
      for (auto i = 0; (i) < (3000); (i) += (1)) {
        auto v = in_[i];
        in[i] = v;
      }
      kernel_hamiltonian<<<blocks, threads, 0, stream>>>(out, in);
      hipStreamSynchronize(stream);
      for (auto i = 0; (i) < (3000); (i) += (1)) {
        auto v = out[i];
        out_[i] = v;
      };
    };
  }
  prob.FindEigenvectors();
  for (auto i = 0; (i) < (3); (i) += (1)) {

    (std::cout) << (std::setw(10))
                << (std::chrono::high_resolution_clock::now()
                        .time_since_epoch()
                        .count())
                << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__)
                << (":") << (__LINE__) << (" ") << (__func__) << (" ") << ("")
                << (" ") << (std::setw(8)) << (" i='") << (i) << ("'")
                << (std::setw(8)) << (" prob.Eigenvalue(i)='")
                << (prob.Eigenvalue(i)) << ("'") << (std::endl) << (std::flush);
  }
  for (auto i = 0; (i) < (1); (i) += (1)) {
    for (auto j = 0; (j) < (3000); (j) += (1)) {
      auto r = (((1.6666667e-2)) * (((j) + (1))));

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("") << (" ") << (std::setw(8)) << (" i='") << (i)
                  << ("'") << (std::setw(8)) << (" r='") << (r) << ("'")
                  << (std::setw(8)) << (" prob.Eigenvector(i, j)='")
                  << (prob.Eigenvector(i, j)) << ("'") << (std::endl)
                  << (std::flush);
    }
  };
  {
    auto res = hipFree(out);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipFree(out)");
    };
  };
  {
    auto res = hipFree(in);
    if (!((hipSuccess) == (res))) {

      (std::cout) << (std::setw(10))
                  << (std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count())
                  << (" ") << (std::this_thread::get_id()) << (" ")
                  << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
                  << (" ") << ("error:") << (" ") << (std::setw(8))
                  << (" hipGetErrorString(res)='") << (hipGetErrorString(res))
                  << ("'") << (std::endl) << (std::flush);
      throw std::runtime_error("hipFree(in)");
    };
  };

  (std::cout)
      << (std::setw(10))
      << (std::chrono::high_resolution_clock::now().time_since_epoch().count())
      << (" ") << (std::this_thread::get_id()) << (" ") << (__FILE__) << (":")
      << (__LINE__) << (" ") << (__func__) << (" ") << ("end main") << (" ")
      << (std::endl) << (std::flush);
  return 0;
};