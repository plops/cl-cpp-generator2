#include "hip/hip_runtime.h"

#include "utils.h"

#include "globals.h"

#include "proto2.h"
;
extern State state;
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/simpleCUFFT/simpleCUFFT.cu
#include <cassert>

#include "/opt/cuda/targets/x86_64-linux/include/hip/hip_runtime.h"
#include "/opt/cuda/targets/x86_64-linux/include/hipfft/hipfft.h"
#include "/opt/cuda/targets/x86_64-linux/include/hipfft/hipfftw.h"

typedef float2 Complex;

void initProcessing() {
  auto n_cuda = 0;
  {
    auto r = hipGetDeviceCount(&n_cuda);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipGetDeviceCount(&n_cuda) => ") << (r)
                << (" '") << (hipGetErrorString(r)) << ("' ") << (" n_cuda=")
                << (n_cuda) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipSetDevice(0);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipSetDevice(0) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
}
void runProcessing(int index) {
  auto p = reinterpret_cast<Complex *>(state._mmap_data);
  auto range = state._range;
  auto h_signal = &(p[((range) * (index))]);
  Complex *d_signal;
  Complex *d_kernel;
  auto memsize = ((sizeof(Complex)) * (range));
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_signal), memsize);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipMalloc(reinterpret_cast<void**>(&d_signal), memsize) => ")
        << (r) << (" '") << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMemcpy(d_signal, h_signal, memsize, hipMemcpyHostToDevice);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMemcpy(d_signal, h_signal, memsize, "
                    "hipMemcpyHostToDevice) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  hipfftHandle plan;
  {
    auto r = hipfftPlan1d(&plan, range, HIPFFT_C2C, 1);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipfftPlan1d(&plan, range, HIPFFT_C2C, 1) => ") << (r)
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD) => ") << (r)
        << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_kernel),
                        ((sizeof(Complex)) * (range)));
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMalloc(reinterpret_cast<void**>(&d_kernel), "
                    "((sizeof(Complex))*(range))) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (std::endl);
    assert((hipSuccess) == (r));
  };
}
void cleanupProcessing(){};