#include "hip/hip_runtime.h"

#include "utils.h"

#include "globals.h"

#include "proto2.h"
;
extern State state;
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/simpleCUFFT/simpleCUFFT.cu
#include <cassert>

#include "/opt/cuda/targets/x86_64-linux/include/hip/hip_runtime.h"
#include "/opt/cuda/targets/x86_64-linux/include/hipfft/hipfft.h"
#include "/opt/cuda/targets/x86_64-linux/include/hipfft/hipfftw.h"

typedef float2 Complex;

static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = ((((a.x) * (b.x))) - (((a.y) * (b.y))));
  c.y = ((((a.x) * (b.y))) + (((a.y) * (b.x))));
  return c;
}
static __global__ void ComplexPointwiseMul(Complex *a, Complex *b, int size) {
  auto numThreads = ((blockDim.x) * (gridDim.x));
  auto threadID = ((((blockIdx.x) * (blockDim.x))) + (threadIdx.x));
  for (int i(threadID); i < size; (i) += (numThreads)) {
    a[i] = ComplexMul(a[i], b[i]);
  };
}
void initProcessing() {
  auto n_cuda = 0;
  {
    auto r = hipGetDeviceCount(&n_cuda);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipGetDeviceCount(&n_cuda) => ") << (r)
                << (" '") << (hipGetErrorString(r)) << ("' ") << (" n_cuda=")
                << (n_cuda) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipSetDevice(0);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipSetDevice(0) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
}
std::complex<float> *runProcessing(int index) {
  auto p = reinterpret_cast<Complex *>(state._mmap_data);
  auto range = state._range;
  auto h_signal = &(p[((range) * (index))]);
  Complex *d_signal;
  Complex *d_kernel;
  auto memsize = ((sizeof(Complex)) * (range));
  static Complex *h_signal2 = static_cast<Complex *>(malloc(memsize));
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_signal), memsize);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipMalloc(reinterpret_cast<void**>(&d_signal), memsize) => ")
        << (r) << (" '") << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMemcpy(d_signal, h_signal, memsize, hipMemcpyHostToDevice);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMemcpy(d_signal, h_signal, memsize, "
                    "hipMemcpyHostToDevice) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  hipfftHandle plan;
  {
    auto r = hipfftPlan1d(&plan, range, HIPFFT_C2C, 1);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipfftPlan1d(&plan, range, HIPFFT_C2C, 1) => ") << (r)
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD) => ") << (r)
        << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_kernel),
                        ((sizeof(Complex)) * (range)));
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMalloc(reinterpret_cast<void**>(&d_kernel), "
                    "((sizeof(Complex))*(range))) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  ComplexPointwiseMul<<<32, 256>>>(d_signal, d_kernel, range);
  {
    auto r = hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_BACKWARD);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_BACKWARD) => ") << (r)
        << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMemcpy(h_signal2, d_signal, memsize, hipMemcpyDeviceToHost);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMemcpy(h_signal2, d_signal, memsize, "
                    "hipMemcpyDeviceToHost) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipfftDestroy(plan);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipfftDestroy(plan) => ") << (r)
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipFree(d_signal);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipFree(d_signal) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipFree(d_kernel);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipFree(d_kernel) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  return reinterpret_cast<std::complex<float> *>(h_signal2);
}
void cleanupProcessing(){};