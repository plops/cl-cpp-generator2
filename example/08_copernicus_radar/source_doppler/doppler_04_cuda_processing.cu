#include "hip/hip_runtime.h"

#include "utils.h"

#include "globals.h"

#include "proto2.h"
;
extern State state;
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/simpleCUFFT/simpleCUFFT.cu
#include <cassert>

#include "/opt/cuda/targets/x86_64-linux/include/hip/hip_runtime.h"
#include "/opt/cuda/targets/x86_64-linux/include/hipfft/hipfft.h"
#include "/opt/cuda/targets/x86_64-linux/include/hipfft/hipfftw.h"

#include "data.h"

#include <complex>

#include <cmath>

// fixme: tx configuration for each pulse is currently always the same. for iw
// datasets i have to figure out how to get the tx configuration rank packets in
// the past.

typedef float2 Complex;

static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = ((((a.x) * (b.x))) - (((a.y) * (b.y))));
  c.y = ((((a.x) * (b.y))) + (((a.y) * (b.x))));
  return c;
}
static __global__ void ComplexPointwiseMul(Complex *a, Complex *b, int size) {
  auto numThreads = ((blockDim.x) * (gridDim.x));
  auto threadID = ((((blockIdx.x) * (blockDim.x))) + (threadIdx.x));
  for (int i(threadID); i < size; (i) += (numThreads)) {
    a[i] = ComplexMul(a[i], b[i]);
  };
}
void initProcessing() {
  auto n_cuda = 0;
  {
    auto r = hipGetDeviceCount(&n_cuda);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipGetDeviceCount(&n_cuda) => ") << (r)
                << (" '") << (hipGetErrorString(r)) << ("' ") << (" n_cuda=")
                << (n_cuda) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipSetDevice(0);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipSetDevice(0) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
}
std::complex<float> *runProcessing(int index) {
  Complex *p = reinterpret_cast<Complex *>(state._mmap_data);
  auto range = state._range;
  Complex *h_signal = &(p[((range) * (index))]);
  Complex *d_signal;
  Complex *d_signal_out;
  Complex *d_kernel;
  Complex *d_kernel_out;
  auto memsize = ((sizeof(Complex)) * (range));
  static Complex *h_signal2 = static_cast<Complex *>(malloc(memsize));
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_signal), memsize);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipMalloc(reinterpret_cast<void**>(&d_signal), memsize) => ")
        << (r) << (" '") << (hipGetErrorString(r)) << ("' ") << (" memsize=")
        << (memsize) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_signal_out), memsize);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipMalloc(reinterpret_cast<void**>(&d_signal_out), memsize) => ")
        << (r) << (" '") << (hipGetErrorString(r)) << ("' ") << (" memsize=")
        << (memsize) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMemcpy(d_signal, h_signal, memsize, hipMemcpyHostToDevice);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMemcpy(d_signal, h_signal, memsize, "
                    "hipMemcpyHostToDevice) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (" memsize=") << (memsize) << (" d_signal=") << (d_signal)
                << (" h_signal=") << (h_signal) << (" p=") << (p)
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  hipfftHandle plan;
  {
    auto r = hipfftPlan1d(&plan, range, HIPFFT_C2C, 1);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipfftPlan1d(&plan, range, HIPFFT_C2C, 1) => ") << (r)
                << (" range=") << (range) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipfftExecC2C(plan, d_signal, d_signal_out, HIPFFT_FORWARD);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipfftExecC2C(plan, d_signal, d_signal_out, HIPFFT_FORWARD) => ")
        << (r) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_kernel), memsize);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipMalloc(reinterpret_cast<void**>(&d_kernel), memsize) => ")
        << (r) << (" '") << (hipGetErrorString(r)) << ("' ") << (" memsize=")
        << (memsize) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMalloc(reinterpret_cast<void **>(&d_kernel_out), memsize);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipMalloc(reinterpret_cast<void**>(&d_kernel_out), memsize) => ")
        << (r) << (" '") << (hipGetErrorString(r)) << ("' ") << (" memsize=")
        << (memsize) << (std::endl);
    assert((hipSuccess) == (r));
  };
  if (!(state._kernel_arg)) {
    state._kernel_arg =
        static_cast<float *>(malloc(((sizeof(float)) * (state._range))));
  };
  auto _txprr = txprr[index];
  auto _txpl = txpl[index];
  auto _txpsf = txpsf[index];
  auto _fdec = fdec[index];
  auto h_kernel = static_cast<std::complex<float> *>(malloc(memsize));
  auto xs_off = 0;
  for (int i(0); xs_off < _txpl; (i)++) {
    const std::complex<float> imag(0, 1);
    auto xs_us = ((i) / (_fdec));
    xs_off = ((xs_us) - ((((5.e-1f)) * (_txpl))) - ((5.e-1f)));
    auto arg =
        ((2) * ((3.1415927e+0f)) *
         (((((xs_off) * (((_txpsf) + ((((5.e-1f)) * (_txpl) * (_txprr))))))) +
           (((((xs_off) * (xs_off))) * ((5.e-1f)) * (_txprr))))));
    auto cplx = std::exp(((imag) * (arg)));
    h_kernel[i] = cplx;
    state._kernel_arg[i] = arg;
  }
  {
    auto r = hipMemcpy(d_kernel, h_kernel, memsize, hipMemcpyHostToDevice);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMemcpy(d_kernel, h_kernel, memsize, "
                    "hipMemcpyHostToDevice) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (" memsize=") << (memsize) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipfftExecC2C(plan, d_kernel, d_kernel_out, HIPFFT_FORWARD);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipfftExecC2C(plan, d_kernel, d_kernel_out, HIPFFT_FORWARD) => ")
        << (r) << (std::endl);
    assert((hipSuccess) == (r));
  };
  free(h_kernel);
  ComplexPointwiseMul<<<128, 1024>>>(d_signal_out, d_kernel_out, range);
  // copy data back
  {
    auto h_signal3 = static_cast<Complex *>(malloc(memsize));
    auto v = reinterpret_cast<std::complex<float> *>(h_signal3);
    {
      auto r =
          hipMemcpy(h_signal3, d_signal_out, memsize, hipMemcpyDeviceToHost);
      (std::cout) << (((std::chrono::high_resolution_clock::now()
                            .time_since_epoch()
                            .count()) -
                       (state._start_time)))
                  << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                  << (__func__)
                  << (" hipMemcpy(h_signal3, d_signal_out, memsize, "
                      "hipMemcpyDeviceToHost) => ")
                  << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                  << (" memsize=") << (memsize) << (std::endl);
      assert((hipSuccess) == (r));
    };
    std::setprecision(3);
    (std::cout) << (std::setw(10))
                << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" ") << ("runProcessing") << (" ")
                << (std::setw(8)) << (" v[0]=") << (v[0]) << (std::setw(8))
                << (" v[1]=") << (v[1]) << (std::setw(8)) << (" v[2]=")
                << (v[2]) << (std::setw(8)) << (" v[3]=") << (v[3])
                << (std::setw(8)) << (" v[4]=") << (v[4]) << (std::endl);
    free(h_signal3);
  };
  {
    auto r = hipfftExecC2C(plan, d_signal_out, d_signal, HIPFFT_BACKWARD);
    (std::cout)
        << (((std::chrono::high_resolution_clock::now()
                  .time_since_epoch()
                  .count()) -
             (state._start_time)))
        << (" ") << (__FILE__) << (":") << (__LINE__) << (" ") << (__func__)
        << (" hipfftExecC2C(plan, d_signal_out, d_signal, HIPFFT_BACKWARD) => ")
        << (r) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipMemcpy(h_signal2, d_signal, memsize, hipMemcpyDeviceToHost);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__)
                << (" hipMemcpy(h_signal2, d_signal, memsize, "
                    "hipMemcpyDeviceToHost) => ")
                << (r) << (" '") << (hipGetErrorString(r)) << ("' ")
                << (" memsize=") << (memsize) << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipfftDestroy(plan);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipfftDestroy(plan) => ") << (r)
                << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipFree(d_signal);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipFree(d_signal) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipFree(d_signal_out);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipFree(d_signal_out) => ") << (r)
                << (" '") << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipFree(d_kernel);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipFree(d_kernel) => ") << (r) << (" '")
                << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  {
    auto r = hipFree(d_kernel_out);
    (std::cout) << (((std::chrono::high_resolution_clock::now()
                          .time_since_epoch()
                          .count()) -
                     (state._start_time)))
                << (" ") << (__FILE__) << (":") << (__LINE__) << (" ")
                << (__func__) << (" hipFree(d_kernel_out) => ") << (r)
                << (" '") << (hipGetErrorString(r)) << ("' ") << (std::endl);
    assert((hipSuccess) == (r));
  };
  return reinterpret_cast<std::complex<float> *>(h_signal2);
}
void cleanupProcessing(){};